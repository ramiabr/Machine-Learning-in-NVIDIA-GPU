/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "support.h"
#include "gpukernel.cu"

extern "C" void gpu_kernel_wrapper(float *input_units,  float *hidden_units, float **input_weights, int inp, int hidden);

extern "C"  void gpu_output_error(float *delta, float *target, float *output, int count, float *err, float *hidden_delta, int hid, float **hidden_weights, float **hidden_units, int out);



void gpu_kernel_wrapper(float *input_units,  float *hidden_units_N, float **input_weights, int inp, int hidden) {

  Timer timer; 
  hipError_t cuda_ret;
 
  float *input_units_d, *hidden_units_d, *input_weights_d; 
  float *input_weights_h; 
 
  input_weights_h = (float*) malloc(sizeof(float) * inp * hidden); 
 /// hidden_units_N = (float*) malloc(sizeof(float) *  hidden); 
 
  for(int i=0; i < inp; i++) {
      for(int j=0; j < hidden; j++) {
          input_weights_h[i*hidden+j] =  input_weights[i][j]; 
          //printf("i=%d, j=%d, %d\n", i, j, (i*hidden+j));
      }
  }

   //input_units[0] = first;

  // Allocate device variables ----------------------------------------------
  hipMalloc((void**) &input_units_d, sizeof(float) * inp); 
  hipMalloc((void**) &hidden_units_d, sizeof(float) * hidden); 
  hipMalloc((void**) &input_weights_d, sizeof(float) * inp * hidden); 

  hipDeviceSynchronize();

  // Copy host variables to device ------------------------------------------
  //printf("Copying data from host to device..."); 
  hipMemcpy(input_units_d, input_units, sizeof(float) *inp, hipMemcpyHostToDevice);
  hipMemcpy(input_weights_d, input_weights_h, sizeof(float) *inp * hidden, hipMemcpyHostToDevice);

  hipDeviceSynchronize();  

  // Launch kernel using standard sgemm interface ---------------------------
  //printf("Launching kernel..."); 
  gpu_bpnn_layerforward(input_units_d, hidden_units_d, input_weights_d, inp, hidden); 

  hipDeviceSynchronize();


   hipMemcpy(hidden_units_N, hidden_units_d, sizeof(float) * hidden, hipMemcpyDeviceToHost);
 
  hipDeviceSynchronize();


   free(input_weights_h); 
 
   hipFree(input_units_d);
   hipFree(hidden_units_d);
   hipFree(input_weights_d);
}


void gpu_output_error (float *output_delta, float *target, float *output, int count, float *err, float *hidden_delta, int hid, float **hidden_weights, float **hidden_units, int out) {

    float *output_delta_d, *target_d, *output_d; 
    float *hidden_weights_d, *hidden_units_d, *hidden_delta_d; 
    float *hidden_weights_h;
  
    hidden_weights_h = (float*) malloc(sizeof(float) * hid); 
    /// hidden_units_N = (float*) malloc(sizeof(float) *  hidden); 
 
    for(int i=0; i < hid; i++) {
          hidden_weights_h[i] =  hidden_weights[i][1]; 
     }
             
    hipMalloc((void**) &output_delta_d, sizeof(float) * count); 
    hipMalloc((void**) &target_d, sizeof(float) * count); 
    hipMalloc((void**) &output_d, sizeof(float) * count); 

    hipMalloc((void**) &hidden_units_d, sizeof(float) * hid); 
    hipMalloc((void**) &hidden_weights_d, sizeof(float) * hid); 
    hipMalloc((void**) &hidden_delta_d, sizeof(float) * hid); 
    hipMalloc((void**) &hidden_delta_d, sizeof(float) * hid); 
 
   hipDeviceSynchronize();

    // Copy host variables to device ------------------------------------------
    //printf("gpu_output_error: Copying data from host to device..."); 
    hipMemcpy(target_d, target, sizeof(float) * count, hipMemcpyHostToDevice);
    hipMemcpy(output_d, output, sizeof(float) * count, hipMemcpyHostToDevice);

    hipDeviceSynchronize();

    hipMemcpy(hidden_units_d, hidden_units, sizeof(float) * hid, hipMemcpyHostToDevice);
    hipMemcpy(hidden_weights_d, hidden_weights_h, sizeof(float) * hid , hipMemcpyHostToDevice);
    //hipMemcpy(prev_d, prev, sizeof(float) * 2 * hid , hipMemcpyHostToDevice);


    // Launch kernel using standard sgemm interface ---------------------------
    //printf("gpu_output_error: Launching kernel..."); 
    gpu_output_error_kernel(output_delta_d, target_d, output_d, count, err);
 
    hipDeviceSynchronize();

    hipMemcpy(output_delta, output_delta_d, sizeof(float) * count, hipMemcpyDeviceToHost);
 
    hipDeviceSynchronize(); 

    gpu_hidden_error_kernel(hidden_delta_d, hid, output_delta_d, out, hidden_weights_d, hidden_units_d);  

    hipDeviceSynchronize(); 

     hipMemcpy(hidden_delta, hidden_delta_d, sizeof(float) * hid, hipMemcpyDeviceToHost);
 
    gpu_weight_adjust(output_delta_d, out, hidden_units_d, hid, hidden_weights_d);
     
    hipDeviceSynchronize(); 
   

     hipMemcpy(hidden_weights_h, hidden_weights_d, sizeof(float) * hid, hipMemcpyDeviceToHost);
 
    hipDeviceSynchronize(); 

     for(int i=0; i < hid; i++) {
          hidden_weights[i][1] =  hidden_weights_h[i];
     }

    hipFree(output_delta_d);
    hipFree(target_d);
    hipFree(output_d);  
    hipFree(hidden_weights_d);
    hipFree(hidden_units_d);
    hipFree(hidden_delta_d);
}

